#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// BSD 3-Clause License
//
// Copyright (c) 2021, NVIDIA Corporation
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
//    contributors may be used to endorse or promote products derived from
//    this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
/////////////////////////////////////////////////////////////////////////////////

#include "matx.h"
#include <cassert>
#include <cstdio>
#include <math.h>
#include <memory>

using namespace matx;

/**
 * MatX uses C++ expression templates to build arithmetic expressions that compile into a lazily-evaluated
 * type for executing on the device. Currently, nvcc cannot see certain optimizations
 * when building the expression tree that would be obvious by looking at the code. Specifically any code reusing
 * the same tensor multiple times appears to the compiler as separate tensors, and it may issue multiple load
 * instructions. While caching helps, this can have a slight performance impact when compared to native CUDA
 * kernels. To work around this problem, complex expressions can be placed in a custom operator by adding some
 * boilerplate code around the original expression. This custom operator can then be used either alone or inside
 * other arithmetic expressions, and only a single load is issues for each tensor.
 *
 * This example uses the Black-Scholes equtation to demonstrate three ways to implement the equation in MatX, and
 * shows the performance difference between them. The three ways are:
 * 1. Using a custom operator
 * 2. Using a lambda function via apply()
 * 3. Using a MatX expression
 *
 * Which method to use depends on the use case, but the lambda function is preferred for simplicity and readability.
 */

/* Custom operator */
template <class I1>
class BlackScholes : public BaseOp<BlackScholes<I1>> {
private:
  I1 V_, S_, K_, r_, T_;

public:
  using matxop = bool;

  BlackScholes(I1 K, I1 V, I1 S, I1 r, I1 T)
      : V_(V), S_(S), K_(K), r_(r), T_(T)  {}

  template <detail::ElementsPerThread EPT>
  __MATX_INLINE__ __MATX_HOST__ __MATX_DEVICE__ auto operator()(index_t idx) const
  {
    auto V = V_(idx);
    auto K = K_(idx);
    auto S = S_(idx);
    auto T = T_(idx);
    auto r = r_(idx);

    auto VsqrtT = V * sqrt(T);
    auto d1 = (log(S / K) + (r + 0.5f * V * V) * T) / VsqrtT ;
    auto d2 = d1 - VsqrtT;
    auto cdf_d1 = normcdff(d1); // Note in a custom op we call the CUDA math function directly
    auto cdf_d2 = normcdff(d2);
    auto expRT = exp(-1.f * r * T);

    return S * cdf_d1 - K * expRT * cdf_d2;
  }

  __MATX_INLINE__ __MATX_HOST__ __MATX_DEVICE__  void operator()(index_t idx) {
    return this->operator()<detail::ElementsPerThread::ONE>(idx);
  }

  __MATX_INLINE__ __MATX_HOST__ __MATX_DEVICE__ index_t Size(uint32_t i) const  { return V_.Size(i); }
  static constexpr __MATX_INLINE__ __MATX_HOST__ __MATX_DEVICE__ int32_t Rank() { return I1::Rank(); }

  template <detail::OperatorCapability Cap>
  __MATX_INLINE__ __MATX_HOST__ auto get_capability() const {  
    // Don't support vectorization yet
    if constexpr (Cap == detail::OperatorCapability::ELEMENTS_PER_THREAD) {
      return detail::ElementsPerThread::ONE;
    } else {    
      auto self_has_cap = detail::capability_attributes<Cap>::default_value;
      return detail::combine_capabilities<Cap>(
          self_has_cap,
        detail::get_operator_capability<Cap>(V_),
        detail::get_operator_capability<Cap>(S_),
        detail::get_operator_capability<Cap>(K_),
        detail::get_operator_capability<Cap>(r_),
        detail::get_operator_capability<Cap>(T_)
      );
    }
  }
};

/* Arithmetic expression */
template<typename T1>
void compute_black_scholes_matx(tensor_t<T1,1>& K,
                                tensor_t<T1,1>& S,
                                tensor_t<T1,1>& V,
                                tensor_t<T1,1>& r,
                                tensor_t<T1,1>& T,
                                tensor_t<T1,1>& output,
                                cudaExecutor& exec)
{
    auto VsqrtT = V * sqrt(T);
    auto d1 = (log(S / K) + (r + 0.5f * V * V) * T) / VsqrtT ;
    auto d2 = d1 - VsqrtT;
    auto cdf_d1 = normcdf(d1);
    auto cdf_d2 = normcdf(d2);
    auto expRT = exp(-1.f * r * T);

    (output = S * cdf_d1 - K * expRT * cdf_d2).run(exec);
}

int main([[maybe_unused]] int argc, [[maybe_unused]] char **argv)
{
  MATX_ENTER_HANDLER();

  using dtype = float;

  index_t input_size = 100000000;
  constexpr uint32_t num_iterations = 100;
  float time_ms;

  tensor_t<dtype, 1> K_tensor{{input_size}};
  tensor_t<dtype, 1> S_tensor{{input_size}};
  tensor_t<dtype, 1> V_tensor{{input_size}};
  tensor_t<dtype, 1> r_tensor{{input_size}};
  tensor_t<dtype, 1> T_tensor{{input_size}};
  tensor_t<dtype, 1> output_tensor{{input_size}};
  tensor_t<dtype, 1> output_tensor2{{input_size}};
  tensor_t<dtype, 1> output_tensor3{{input_size}};

  (K_tensor = random<float>({input_size}, UNIFORM)).run();
  (S_tensor = random<float>({input_size}, UNIFORM)).run();
  (V_tensor = random<float>({input_size}, UNIFORM)).run();
  (r_tensor = random<float>({input_size}, UNIFORM)).run();
  (T_tensor = random<float>({input_size}, UNIFORM)).run();

  hipStream_t stream;
  hipStreamCreate(&stream);
  cudaExecutor exec{stream};

  //compute_black_scholes_matx(K_tensor, S_tensor, V_tensor, r_tensor, T_tensor, output_tensor, exec);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, stream);
  // Time non-operator version
  for (uint32_t i = 0; i < num_iterations; i++) {
    compute_black_scholes_matx(K_tensor, S_tensor, V_tensor, r_tensor, T_tensor, output_tensor, exec);
  }
  hipEventRecord(stop, stream);
  exec.sync();

  hipEventElapsedTime(&time_ms, start, stop);

  printf("Time without custom operator = %.2fms per iteration\n",
         time_ms / num_iterations);

  hipEventRecord(start, stream);
  // Time non-operator version
  for (uint32_t i = 0; i < num_iterations; i++) {
    (output_tensor2 = BlackScholes(K_tensor, V_tensor, S_tensor, r_tensor, T_tensor)).run(exec);
  }
  hipEventRecord(stop, stream);
  exec.sync();

  hipEventElapsedTime(&time_ms, start, stop);
  printf("Time with custom operator = %.2fms per iteration\n",
    time_ms / num_iterations);

  auto bs_lambda = [] __device__ (auto K,
                                  auto S,
                                  auto V,
                                  auto r,
                                  auto T) {
      auto VsqrtT = V * sqrt(T);
      auto d1 = (log(S / K) + (r + 0.5f * V * V) * T) / VsqrtT ;
      auto d2 = d1 - VsqrtT;
      auto cdf_d1 = normcdf(d1);
      auto cdf_d2 = normcdf(d2);
      auto expRT = exp(-1.f * r * T);
  
      return S * cdf_d1 - K * expRT * cdf_d2; 
  };

  hipEventRecord(start, stream);
  for (uint32_t i = 0; i < num_iterations; i++) {
    (output_tensor3 = matx::apply(bs_lambda, K_tensor, S_tensor, V_tensor, r_tensor, T_tensor)).run(exec);
  }
  
  hipEventRecord(stop, stream);
  exec.sync();

  hipEventElapsedTime(&time_ms, start, stop);
  printf("Time with lambda = %.2fms per iteration\n",
    time_ms / num_iterations);

  // Verify all 3 outputs match within 1e-6 using operator() (Managed Memory)
  bool all_match = true;
  constexpr float tol = 1e-6f;
  auto n = K_tensor.Size(0);

  for (index_t i = 0; i < n; i++) {
    float v1 = output_tensor(i);
    float v2 = output_tensor2(i);
    float v3 = output_tensor3(i);
    if (fabsf(v1 - v2) > tol || fabsf(v1 - v3) > tol || fabsf(v2 - v3) > tol) {
      printf("Mismatch at idx %lld: v1=%.8f v2=%.8f v3=%.8f\n", i, v1, v2, v3);
      all_match = false;
      break;
    }
  }
  if (all_match) {
    printf("All outputs match within %.1e tolerance.\n", tol);
  } else {
    printf("Outputs do NOT match within %.1e tolerance!\n", tol);
  }

  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipStreamDestroy(stream);
  MATX_CUDA_CHECK_LAST_ERROR();
  MATX_EXIT_HANDLER();
}
